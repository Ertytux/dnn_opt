#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/transform.h>
#include <thrust/reduce.h>
#include <cuda/base/solution.h>

using namespace thrust;

namespace dnn_opt
{
namespace cuda
{

namespace ops
{
namespace solution
{

struct constrains : public thrust::unary_function<float, float>
{
  const float _min;
  const float _max;

  constrains(float min, float max) : _min(min), _max(max)
  {

  }

  __host__ __device__
  float operator()(const float& param) const
  {
    return fminf(_max, fmaxf(_min, param));
  }
};

} // namespace solution
} // namespace ops

generator* solution::get_generator() const
{
  return _dev_generator;
}

void solution::set_constrains()
{
  float min = get_generator()->get_min();
  float max = get_generator()->get_max();
  auto ptr = thrust::device_pointer_cast(get_params());

  transform(ptr, ptr + size(), ptr, ops::solution::constrains(min, max));

  set_modified(true);
}

void solution::init()
{
  hipFree(_params);
  hipMalloc(&_params, size() * sizeof(float));

  _evaluations = 0;
  _modified = true;
}

solution::solution(generator* generator, unsigned int size)
: core::solution(generator, size)
{
  _dev_generator = generator;
}

solution::~solution()
{
  hipFree(_params);

  /* avoid double free from core::solution destructor */
  _params = 0;
}

} // namespace cuda
} // namespace dnn_opt
